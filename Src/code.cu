#include "hip/hip_runtime.h"
% % writefile matrix_multiplication.cu

#include <stdio.h>
#include <stdlib.h>

    // Function to initialize a matrix with random values
    void
    initMatrix(float *matrix, int size)
{
    for (int i = 0; i < size; i++)
    {
        matrix[i] = (float)(rand() % 100);
    }
}

// CPU matrix multiplication
void matrixMulCPU(float *A, float *B, float *C, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            float sum = 0.0;
            for (int k = 0; k < n; k++)
            {
                sum += A[i * n + k] * B[k * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// GPU matrix multiplication - 2D kernel
__global__ void matrixMulGPU_2D(float *A, float *B, float *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    if (row < n && col < n)
    {
        float sum = 0.0;
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

// GPU matrix multiplication - 1D kernel
__global__ void matrixMulGPU_1D(float *A, float *B, float *C, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Linear thread index
    int row = idx / n;                               // Calculate row from index
    int col = idx % n;                               // Calculate column from index

    if (row < n && col < n)
    {
        float sum = 0.0;
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Function to compare CPU and GPU results
void verifyResults(float *C_CPU, float *C_GPU, int n)
{
    for (int i = 0; i < n * n; i++)
    {
        if (fabs(C_CPU[i] - C_GPU[i]) > 1e-4)
        {
            printf("Mismatch at index %d: CPU = %f, GPU = %f\n", i, C_CPU[i], C_GPU[i]);
            return;
        }
    }
    printf("Results are correct!\n");
}

int main()
{
    srand(time(NULL));

    int sizes[] = {256, 512}; // Different matrix sizes to test
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

    for (int test = 0; test < num_sizes; test++)
    {
        int N = sizes[test];
        printf("\nMatrix Size: %d x %d\n", N, N);

        int size = N * N * sizeof(float);

        // Allocate host memory
        float *h_A = (float *)malloc(size);
        float *h_B = (float *)malloc(size);
        float *h_C_CPU = (float *)malloc(size);
        float *h_C1_GPU = (float *)malloc(size);
        float *h_C2_GPU = (float *)malloc(size);

        // Initialize matrices
        initMatrix(h_A, N * N);
        initMatrix(h_B, N * N);

        // CUDA Events for CPU Timing
        hipEvent_t startCPU, endCPU;
        hipEventCreate(&startCPU);
        hipEventCreate(&endCPU);

        hipEventRecord(startCPU);
        matrixMulCPU(h_A, h_B, h_C_CPU, N);
        hipEventRecord(endCPU);
        hipEventSynchronize(endCPU);
        hipDeviceSynchronize();

        float cpu_time;
        hipEventElapsedTime(&cpu_time, startCPU, endCPU);
        printf("CPU Execution Time: %f ms\n", cpu_time);
        printf("\n");

        // Allocate device memory
        float *d_A, *d_B, *d_C1, *d_C2;
        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
        hipMalloc((void **)&d_C1, size);
        hipMalloc((void **)&d_C2, size);

        // Data transfer and kernel timing for 1D kernel
        hipEvent_t startHtoD1D, endHtoD1D, startKernel1D, endKernel1D, startDtoH1D, endDtoH1D;
        hipEventCreate(&startHtoD1D);
        hipEventCreate(&endHtoD1D);
        hipEventCreate(&startKernel1D);
        hipEventCreate(&endKernel1D);
        hipEventCreate(&startDtoH1D);
        hipEventCreate(&endDtoH1D);

        // Host to Device Transfer (1D Kernel)
        hipEventRecord(startHtoD1D);
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        hipEventRecord(endHtoD1D);
        hipEventSynchronize(endHtoD1D);
        hipDeviceSynchronize();

        float htoD_time_1D;
        hipEventElapsedTime(&htoD_time_1D, startHtoD1D, endHtoD1D);

        // 1D Kernel Execution
        int threads_per_block_1D = 256;
        int num_blocks_1D = (N * N + threads_per_block_1D - 1) / threads_per_block_1D;

        hipEventRecord(startKernel1D);
        matrixMulGPU_1D<<<num_blocks_1D, threads_per_block_1D>>>(d_A, d_B, d_C1, N);
        hipEventRecord(endKernel1D);
        hipEventSynchronize(endKernel1D);
        hipDeviceSynchronize();

        float kernel_time_1D;
        hipEventElapsedTime(&kernel_time_1D, startKernel1D, endKernel1D);

        // Device to Host Transfer (1D Kernel)
        hipEventRecord(startDtoH1D);
        hipMemcpy(h_C1_GPU, d_C1, size, hipMemcpyDeviceToHost);
        hipEventRecord(endDtoH1D);
        hipEventSynchronize(endDtoH1D);
        hipDeviceSynchronize();

        float dtoH_time_1D;
        hipEventElapsedTime(&dtoH_time_1D, startDtoH1D, endDtoH1D);

        float total_transfer_time_1D = htoD_time_1D + dtoH_time_1D;
        float total_time_1D = total_transfer_time_1D + kernel_time_1D;
        printf("GPU Execution Time (1D Kernel): %f ms\n", kernel_time_1D);
        printf("GPU Data Transfer Time (HtoD, 1D Kernel): %f ms\n", htoD_time_1D);
        printf("GPU Data Transfer Time (DtoH, 1D Kernel): %f ms\n", dtoH_time_1D);
        printf("Total Data Transfer Time (1D Kernel): %f ms\n", total_transfer_time_1D);
        printf("Total Time (1D Kernel): %f ms\n", total_time_1D);
        verifyResults(h_C_CPU, h_C1_GPU, N);
        printf("\n");

        // Data transfer and kernel timing for 2D kernel
        hipEvent_t startHtoD2D, endHtoD2D, startKernel2D, endKernel2D, startDtoH2D, endDtoH2D;
        hipEventCreate(&startHtoD2D);
        hipEventCreate(&endHtoD2D);
        hipEventCreate(&startKernel2D);
        hipEventCreate(&endKernel2D);
        hipEventCreate(&startDtoH2D);
        hipEventCreate(&endDtoH2D);

        // Host to Device Transfer (2D Kernel)
        hipEventRecord(startHtoD2D);
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        hipEventRecord(endHtoD2D);
        hipEventSynchronize(endHtoD2D);
        hipDeviceSynchronize();

        float htoD_time_2D;
        hipEventElapsedTime(&htoD_time_2D, startHtoD2D, endHtoD2D);

        // 2D Kernel Execution
        dim3 threads_per_block_2D(16, 16);
        dim3 num_blocks_2D((N + 16 - 1) / 16, (N + 16 - 1) / 16);

        hipEventRecord(startKernel2D);
        matrixMulGPU_2D<<<num_blocks_2D, threads_per_block_2D>>>(d_A, d_B, d_C2, N);
        hipEventRecord(endKernel2D);
        hipEventSynchronize(endKernel2D);
        hipDeviceSynchronize();

        float kernel_time_2D;
        hipEventElapsedTime(&kernel_time_2D, startKernel2D, endKernel2D);

        // Device to Host Transfer (2D Kernel)
        hipEventRecord(startDtoH2D);
        hipMemcpy(h_C2_GPU, d_C2, size, hipMemcpyDeviceToHost);
        hipEventRecord(endDtoH2D);
        hipEventSynchronize(endDtoH2D);
        hipDeviceSynchronize();

        float dtoH_time_2D;
        hipEventElapsedTime(&dtoH_time_2D, startDtoH2D, endDtoH2D);

        float total_transfer_time_2D = htoD_time_2D + dtoH_time_2D;
        float total_time_2D = total_transfer_time_2D + kernel_time_2D;
        printf("GPU Execution Time (2D Kernel): %f ms\n", kernel_time_2D);
        printf("GPU Data Transfer Time (HtoD, 2D Kernel): %f ms\n", htoD_time_2D);
        printf("GPU Data Transfer Time (DtoH, 2D Kernel): %f ms\n", dtoH_time_2D);
        printf("Total Data Transfer Time (2D Kernel): %f ms\n", total_transfer_time_2D);
        printf("Total Time (2D Kernel): %f ms\n", total_time_2D);
        verifyResults(h_C_CPU, h_C2_GPU, N);
        printf("\n");

        // Cleanup
        free(h_A);
        free(h_B);
        free(h_C_CPU);
        free(h_C1_GPU);
        free(h_C2_GPU);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C1);
        hipFree(d_C2);

        hipEventDestroy(startCPU);
        hipEventDestroy(endCPU);
        hipEventDestroy(startHtoD1D);
        hipEventDestroy(endHtoD1D);
        hipEventDestroy(startKernel1D);
        hipEventDestroy(endKernel1D);
        hipEventDestroy(startDtoH1D);
        hipEventDestroy(endDtoH1D);
        hipEventDestroy(startHtoD2D);
        hipEventDestroy(endHtoD2D);
        hipEventDestroy(startKernel2D);
        hipEventDestroy(endKernel2D);
        hipEventDestroy(startDtoH2D);
        hipEventDestroy(endDtoH2D);
    }

    return 0;
}
